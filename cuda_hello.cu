#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "iostream"
__global__ void sayHelloWorld();

void main(void) {

	printf("HelloWorld! CPU \n");
	sayHelloWorld << <1, 6>> > ();   //调用GPU上执行的函数，调用GPU线程
	hipDeviceReset();    //显式地释放和清空当前进程中与当前设备有关的所有资源，不加这句不会打印GPU中的输出语句"HelloWorld! GPU"
  system("pause");

}

__global__ void sayHelloWorld() {
	printf("HelloWorld! GPU \n");
}
